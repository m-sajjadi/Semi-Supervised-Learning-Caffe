#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/loss_ts_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void Forward_ts(const int nthreads, const Dtype* in_data,
    const int nt, const int dim, Dtype* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {        
    const int ind1 = (nt*dim)*(index /(nt*nt*dim)) + index % (nt*dim);
    const int ind2 = dim*(index /(dim*nt)) + index % dim;    
    out_data[index] = in_data[ind1] - in_data[ind2];    
  }
}

template <typename Dtype>
void LossTSLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
    
  const Dtype* dt = bottom[0]->gpu_data();
  Dtype* tmp1_ = tmp1.mutable_gpu_data();
  Dtype* tmp2_ = tmp2.mutable_gpu_data();  
  const int nthreads = tmp1.count();
  Forward_ts<Dtype>
        <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
        nthreads, dt, nt, shp[1], tmp1_);
    
  caffe_gpu_powx<Dtype>(nthreads, tmp1_, Dtype(2), tmp2_); 
  Dtype loss;
  caffe_gpu_asum(nthreads, tmp2_, &loss); 
  loss = loss/shp[0];

  top[0]->mutable_cpu_data()[0] = loss;
  

}

template <typename Dtype>
__global__ void Backward_ts(const int nthreads, const Dtype* in_data,
    const int nt, const int dim, Dtype* out_data, const Dtype lambda) {
  CUDA_KERNEL_LOOP(index, nthreads) {        
    const int ind1 = (nt*dim)*(index /dim) + index % dim;    
    out_data[index] = 0;
    for (int i=0; i<nt; i++){
      out_data[index] = out_data[index] - lambda*in_data[ind1 + dim*i];
    }
  }
}

template <typename Dtype>
void LossTSLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    
    const Dtype* tmp1_ = tmp1.gpu_data();     
    const int nthreads = bottom[0]->count();
    
    Backward_ts<Dtype>
	  <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
	  nthreads, tmp1_, nt, shp[1], bottom_diff, lambda); 	  
  }	
  
}

INSTANTIATE_LAYER_GPU_FUNCS(LossTSLayer);

}  // namespace caffe
