#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/loss_mx_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void Forward_tx(const int nthreads, Dtype* in_data,
    const int dim, Dtype* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {      
    const int ind = dim*index;  
    float p = 1;
    
    Dtype ep = FLT_EPSILON;    
    
    for (int i = 0; i < dim; i++){
      
      in_data[ind+i] = max(in_data[ind+i], ep);
      in_data[ind+i] = min(in_data[ind+i], 1-ep);
      
      p = p * (1 - in_data[ind + i]);
    }
    out_data[index] = 0;    
    for (int i = 0; i < dim; i++){
      out_data[index] = out_data[index] + p * in_data[ind + i] / (1 - in_data[ind + i]);
    }
  }
}

template <typename Dtype>
void LossMXLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
    
  Dtype* dt = bottom[0]->mutable_gpu_data();
  Dtype* tmp1_ = tmp1.mutable_gpu_data();
  const int nthreads = tmp1.count();  
  Forward_tx<Dtype>
        <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
        nthreads, dt, shp[1], tmp1_);
  
  Dtype loss;
  caffe_gpu_asum(nthreads, tmp1_, &loss);  
  loss = loss/shp[0];

  top[0]->mutable_cpu_data()[0] = loss;
  
}

template <typename Dtype>
__global__ void Backward_tx(const int nthreads, const Dtype* in_data,
    const int dim, Dtype* out_data, const Dtype lambda) {
  CUDA_KERNEL_LOOP(index, nthreads) {    
    
    const int ind = (index / dim) * dim;
    float p = 1;
    for (int i = 0; i<dim; i++){
	p = p * (1 - in_data[ ind + i ]);
    }		    

    float t1 = p/(1 - in_data[index]);	
    
    float t2 = 0;
    for (int i = 0; i<dim; i++){
	t2 = t2 + p * in_data[ ind + i ] / (1 - in_data[ ind + i ]) / (1 - in_data[ index ]);	  
    }
    t2 = -t2;
    float t3 = p*in_data[index]/(1 - in_data[index])/(1 - in_data[index]);
    
    float v = (t1 + t2 + t3);    
    
    out_data[index] = -lambda*v;
  }
}

template <typename Dtype>
void LossMXLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    
    const Dtype* dt = bottom[0]->gpu_data();
    const int nthreads = bottom[0]->count();    
    Backward_tx<Dtype>
	  <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
	  nthreads, dt, shp[1], bottom_diff, lambda);    
  }	
  
}

INSTANTIATE_LAYER_GPU_FUNCS(LossMXLayer);

}  // namespace caffe
